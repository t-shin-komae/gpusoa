#include "array.hpp"
#include "cuda_utils.hpp"
#include <stdexcept>

namespace soa {
KArray1D::KArray1D(double *D_ptr, int Nx) : D_ptr(D_ptr), Nx(Nx) {}
#ifdef __HIPCC__
Array1D::Array1D(int Nx) : Nx(Nx) {
  std::cout << "hipMalloc\n" << std::endl;
  check(hipHostMalloc(&H_ptr, sizeof(double) * Nx));
  check(hipMalloc(&D_ptr, sizeof(double) * Nx));
}
Array1D::Array1D(int Nx, std::function<double(int)> f) : Array1D(Nx) {
  for (int i = 0; i < Nx; i++)
    H_ptr[i] = f(i);

  check(hipMemcpy(D_ptr, H_ptr, sizeof(double) * Nx, hipMemcpyHostToDevice));
}

Array1D::~Array1D() {
  std::cout << "hipFree\n" << std::endl;
  check(hipHostFree(H_ptr));
  check(hipFree(D_ptr));
}
void Array1D::DeviceToHost() {
  check(hipMemcpy(H_ptr, D_ptr, sizeof(double) * Nx, hipMemcpyDeviceToHost));
}
void Array1D::HostToDevice() {
  check(hipMemcpy(D_ptr, H_ptr, sizeof(double) * Nx, hipMemcpyHostToDevice));
}

#endif
} // namespace soa
