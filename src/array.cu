#include "array.hpp"
#include "cuda_utils.hpp"
#include <stdexcept>

namespace soa {
void Array1D::DeviceToHost() {
  check(hipMemcpy(H_ptr, D_ptr, sizeof(double) * Nx, hipMemcpyDeviceToHost));
}
void Array1D::HostToDevice() {
  check(hipMemcpy(D_ptr, H_ptr, sizeof(double) * Nx, hipMemcpyHostToDevice));
}
void Array2D::DeviceToHost() {
  check(hipMemcpy(H_ptr, D_ptr, sizeof(double) * Nx * Ny,
                   hipMemcpyDeviceToHost));
}
void Array2D::HostToDevice() {
  check(hipMemcpy(D_ptr, H_ptr, sizeof(double) * Nx * Ny,
                   hipMemcpyHostToDevice));
}
void Array3D::DeviceToHost() {
  check(hipMemcpy(H_ptr, D_ptr, sizeof(double) * Nx * Ny * Nz,
                   hipMemcpyDeviceToHost));
}
void Array3D::HostToDevice() {
  check(hipMemcpy(D_ptr, H_ptr, sizeof(double) * Nx * Ny * Nz,
                   hipMemcpyHostToDevice));
}
} // namespace soa
